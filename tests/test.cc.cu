#include "hip/hip_runtime.h"

#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdexcept>

using namespace std;

class CudaException : public std::runtime_error {
 public:
  CudaException(const std::string& what) : runtime_error(what) {}
};

inline void cuda_check_(hipError_t val, const char* file, int line) {
  if (val != hipSuccess) {
    std::ostringstream os;
    os << file << ':' << line << ": CUDA error " << hipGetErrorName(val)
       << " (#" << val << "): " << hipGetErrorString(val);
    throw CudaException(os.str());
  }
}

#define CUDA_CHECK(val)                     \
  do {                                      \
    cuda_check_((val), __FILE__, __LINE__); \
  } while (0)

struct __align__(16) Bucket {
  uint64_t* keys;      // ignore it!
  uint64_t* metas;     // ignore it!
  ValueType* cache;    // ignore it!
  ValueType* vectors;  // <<<----important member
  uint64_t cur_meta;   // ignore it!
  uint64_t min_meta;   // ignore it!
  int min_pos;         // ignore it!
};

__global__ void write_read(Bucket* buckets, int bucket_idx,
                           const ValueType val) {
  size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  ValueType* vectors = buckets[bucket_idx].vectors;
  *(vectors + tid * DIM) = val;
}

__global__ void read_when_error(Bucket* buckets, int bucket_idx,
                                int vector_idx) {
  ValueType* vectors = buckets[bucket_idx].vectors;
  ValueType val = *(vectors + vector_idx * DIM);
  printf("device view: ptr=%p\tval=%d\n", (vectors + vector_idx * DIM), val);
}

using ValueType = int;

constexpr size_t DIM = 16;
constexpr size_t num_vector = 8 * 16777216;
constexpr size_t num_vector_per_bucket = 128;
constexpr size_t num_buckets = num_vector / num_vector_per_bucket;

constexpr size_t memory_pool_size = num_vector * sizeof(ValueType) * DIM;
constexpr size_t bucket_size = num_vector_per_bucket * sizeof(ValueType) * DIM;


int main() {
  Bucket* buckets;
  hipMallocManaged(&buckets, sizeof(Bucket) * num_buckets);

  std::cout << "size of Bucket=" << sizeof(Bucket) << std::endl;

  assert(num_buckets == (1024 * 1024));
  assert(memory_pool_size == (8ul << 30));
  assert(bucket_size == (128 * 4 * 16));
  assert(memory_pool_size == (bucket_size * num_buckets));

  ValueType* host_memory_pool;
  CUDA_CHECK(hipHostAlloc(&host_memory_pool, memory_pool_size,
                           hipHostMallocMapped | hipHostMallocWriteCombined));

  for (int i = 0; i < num_buckets; i++) {
    ValueType* h_memory_pool = host_memory_pool + (num_vector_per_bucket * DIM * i);
    CUDA_CHECK(hipHostGetDevicePointer(&(buckets[i].vectors), h_memory_pool, 0));
  }
  std::cout << "finish allocating"
            << ", num_buckets=" << num_buckets << ", memory_pool_size=" << (8ul << 30)
            << ", bucket_size=" << (128 * 4 * 16) << std::endl;

  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  ValueType magic_numbers = 88;
  for (int i = 0; i < num_buckets; i++) {
    write_read<<<1, num_vector_per_bucket, 0, stream>>>(buckets, i,
                                                        magic_numbers);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
  CUDA_CHECK(hipDeviceSynchronize());
  std::cout << "finish writing" << std::endl;

  size_t error_num = 0;
  size_t correct_num = 0;
  for (int i = 0; i < num_buckets; i++) {
    for (int j = 0; j < num_vector_per_bucket; j++) {
      ValueType val = host_memory_pool[i * num_vector_per_bucket * DIM + j * DIM];
      if (val != magic_numbers) {
        read_when_error<<<1, 1, 0, stream>>>(buckets, i, j);
        CUDA_CHECK(hipStreamSynchronize(stream));
        printf("host   view: ptr=%p\tval=%d\n\n",
               (host_memory_pool + i * num_vector_per_bucket * DIM + j * DIM), val);
        error_num++;
      } else {
        correct_num++;
      }
    }
  }
  std::cout << "error_num=" << error_num << "\tcorrect_num=" << correct_num
            << std::endl;

  CUDA_CHECK(hipStreamSynchronize(stream));
  CUDA_CHECK(hipStreamDestroy(stream));
  std::cout << "finish checking" << std::endl;

  CUDA_CHECK(hipHostFree(host_memory_pool));
  CUDA_CHECK(hipFree(buckets));
}