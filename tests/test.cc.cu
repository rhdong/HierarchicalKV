#include "hip/hip_runtime.h"

#include <iostream>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <cuda/std/semaphore>

#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <limits>
#include <sstream>
#include <stdexcept>

using namespace cooperative_groups;
namespace cg = cooperative_groups;
using namespace std;


using K = uint64_t;
using M = uint64_t;
using V = int;


constexpr size_t DIM = 16;
constexpr size_t num_vectors_per_slice = 8 * 16777216;
constexpr size_t num_vector_per_bucket = 128;
constexpr size_t TILE_SIZE = 4;

template <class V, uint32_t TILE_SIZE = 4>
__device__ __forceinline__ void copy_vector(
    cg::thread_block_tile<TILE_SIZE> const& g, const V val, V* dst,
    const size_t dim) {
  for (auto i = g.thread_rank(); i < dim; i += g.size()) {
    dst[i] = val;
  }
}

template <class K, class V, class M, uint32_t TILE_SIZE = 4>
__global__ void upsert_kernel_with_io_core(V** slices, int* index, size_t N) {

  size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  auto g = cg::tiled_partition<TILE_SIZE>(cg::this_thread_block());
  int rank = g.thread_rank();


  for (size_t t = tid; t < N; t += blockDim.x * gridDim.x) {
    size_t vector_idx = index[t / TILE_SIZE];
    const V val= static_cast<V>(vector_idx * 0.00001);
    size_t target_slice = vector_idx / num_vectors_per_slice;
    size_t target_offset = (vector_idx % num_vectors_per_slice) * DIM;
    copy_vector<V, TILE_SIZE>(g, val, slices[target_slice] + target_offset, DIM);
  }

};

static inline size_t SAFE_GET_GRID_SIZE(size_t N, int block_size) {
  return  (((N)-1) / block_size + 1);
};

class CudaException : public std::runtime_error {
 public:
  CudaException(const std::string& what) : runtime_error(what) {}
};

inline void cuda_check_(hipError_t val, const char* file, int line) {
  if (val != hipSuccess) {
    std::ostringstream os;
    os << file << ':' << line << ": CUDA error " << hipGetErrorName(val)
       << " (#" << val << "): " << hipGetErrorString(val);
    throw CudaException(os.str());
  }
}

#define CUDA_CHECK(val)                                 \
  do {                                                  \
    cuda_check_((val), __FILE__, __LINE__); \
  } while (0)


template <class K>
using AtomicKey = cuda::atomic<K, cuda::thread_scope_device>;

template <class M>
using AtomicMeta = cuda::atomic<M, cuda::thread_scope_device>;

template <class T>
using AtomicPos = cuda::atomic<T, cuda::thread_scope_device>;


template <class K, class V, class M>
struct Bucket {
  AtomicKey<K>* keys;    // HBM
  AtomicMeta<M>* metas;  // HBM
  V* cache;              // HBM(optional)
  V* vectors;            // Pinned memory or HBM

  /* For upsert_kernel without user specified metas
     recording the current meta, the cur_meta will
     increment by 1 when a new inserting happens. */
  AtomicMeta<M> cur_meta;

  /* min_meta and min_pos is for or upsert_kernel
     with user specified meta. They record the minimum
     meta and its pos in the bucket. */
  AtomicMeta<M> min_meta;
  AtomicPos<int> min_pos;
};

template <class K, class V, class M>
__global__ void write_read(Bucket<K, V, M>* buckets, int bucket_idx, int vector_idx, const V val) {
  V* vectors = buckets[bucket_idx].vectors;
  *(vectors + vector_idx * DIM) = val;
}

int main() {
  int num_slices = 1;
  V** slices;
  size_t slice_size = num_vectors_per_slice * sizeof(V) * DIM;
  hipMallocManaged(&slices, sizeof(V*) * num_slices);

  int num_buckets = num_vectors_per_slice / num_vector_per_bucket;
  Bucket<K, V, M>* buckets;
  size_t bucket_size = num_vector_per_bucket * sizeof(V) * DIM;
  hipMallocManaged(&buckets, sizeof(Bucket<K, V, M>*) * num_buckets);

  assert(num_buckets == (1024 * 1024));
  assert(slice_size == (8ul << 30));
  assert(bucket_size == (128 * 4 * 16));

//  int* d_index;
  V* slice;
//  hipMalloc(&d_index, num_vectors_per_slice * num_slices * sizeof(int));
  hipHostMalloc(&slice, slice_size, hipHostMallocMapped | hipHostMallocWriteCombined);
  slices[0] = slice;

  for(int i = 0; i < num_buckets; i++){
    buckets[i].vectors = slices[0] + (num_vector_per_bucket * DIM * i);
  }
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));
  int magic_numbers = 88;
  for(int i = 0; i < num_buckets; i++){
    for(int j = 0; j < num_vector_per_bucket; j++){
      write_read<K, V, M><<<1, 1, 0, stream>>>(buckets, i, j, magic_numbers);
      CUDA_CHECK(hipStreamSynchronize(stream));
    }
  }

  for(int i = 0; i < num_buckets; i++){
    for(int j = 0; j < num_vector_per_bucket; j++){
      assert(buckets[i].vectors[j * DIM] == magic_numbers);
    }
  }
  CUDA_CHECK(hipStreamSynchronize(stream));
  CUDA_CHECK(hipStreamDestroy(stream));

//  for(int i = 0; i <  num_vectors_per_slice * num_slices; i++){
//    h_index[i] = i;
//  }
//  hipMemcpy(d_index, h_index, num_vectors_per_slice * num_slices * sizeof(int), hipMemcpyHostToDevice);

//  thrust::default_random_engine g;
//  thrust::shuffle(thrust::device, d_index, d_index + num_vectors_per_slice * num_slices, g);


//  const size_t block_size = 128;
//  const size_t N = num_slices * num_vectors_per_slice * TILE_SIZE;
//  const size_t grid_size = SAFE_GET_GRID_SIZE(N, block_size);
//
//  upsert_kernel_with_io_core<K, V, M, 4><<<grid_size, block_size, 0, 0>>>(slices, d_index, N);
//  CUDA_CHECK(hipDeviceSynchronize());
//
//  for(int i = 0; i < num_slices; i++){
//    V* slice = slices[i];
//    for(int j = 0; j < num_vectors_per_slice; j++){
//      float expected = static_cast<V>((i * num_vectors_per_slice + j) * 0.00001);
//      if(expected != slice[j * DIM]){
//        std::cout << expected << " " << slice[j * DIM] << std::endl;
//      }
//    }
//  }

  for(int i = 0; i < num_slices; i++){
    hipHostFree(slices[i]);
  }
  hipFree(slices);
  hipFree(buckets);
//  hipFree(d_index);
//  hipHostFree(h_index);
}